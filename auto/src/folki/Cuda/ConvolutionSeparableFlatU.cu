#include "hip/hip_runtime.h"
/*
      This file is part of FolkiGpu.

    FolkiGpu is free software: you can redistribute it and/or modify
    it under the terms of the GNU Leeser General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Foobar is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Leeser General Public License
    along with FolkiGpu.  If not, see <http://www.gnu.org/licenses/>.

*/

/*
      FolkiGpu is a demonstration software developed by Aurelien Plyer during
    his phd at Onera (2008-2011). For more information please visit :
      - http://www.onera.fr/dtim-en/gpu-for-image/folkigpu.php
      - http://www.plyer.fr (author homepage)
*/

#include "ConvolutionSeparableFlatU.hpp"
#define UNROLL_INNER

template<int i> __device__ float convolutionRowU(float *data){
	return
		data[U_KERNEL_RADIUS - i]
		+ convolutionRowU<i - 1>(data);
}

template<> __device__ float convolutionRowU<0>(float *data){
	return data[U_KERNEL_RADIUS];
}

template<int i> __device__ float convolutionColumnU(float *data){
	return 
		data[(U_KERNEL_RADIUS - i) * COL_W] 
		+ convolutionColumnU<i - 1>(data);
}

template<> __device__ float convolutionColumnU<0>(float *data){
	return data[U_KERNEL_RADIUS* COL_W];
}

/* ======================================== kernels ======================================== */
__global__ void convolutionRowGPUFlatU(
	float *d_Result,
	float *d_Data,
	unsigned int kernelRadius,
	int3 imSize)
{
	extern __shared__ float data[];
	const int         tileStart = IMUL(blockIdx.x, ROW_W);
	const int           tileEnd = tileStart + ROW_W - 1;
	const int        apronStart = tileStart - kernelRadius;
	const int          apronEnd = tileEnd   + kernelRadius;
	const int    tileEndClamped = min(tileEnd, imSize.x - 1);
	const int apronStartClamped = max(apronStart, 0);
	const int   apronEndClamped = min(apronEnd, imSize.x - 1);
	const int          rowStart = IMUL(blockIdx.y, imSize.z);
	const int apronStartAligned = tileStart - iAlignUp(kernelRadius,ALLIGN_ROW);
	const int           loadPos = apronStartAligned + threadIdx.x;
	const int          writePos = tileStart + threadIdx.x;

#ifndef UNROLL_INNER
	float sum ;
#endif

	if(loadPos >= apronStart){
		const int smemPos = loadPos - apronStart;
		data[smemPos] = 
			((loadPos >= apronStartClamped) && (loadPos <= apronEndClamped)) ?
			d_Data[rowStart + loadPos] : 0;
	}
	__syncthreads();
	if(writePos <= tileEndClamped){
		const int smemPos = writePos - apronStart;
#ifndef UNROLL_INNER
		sum=0;
		for(int k = 0; k < kernelRadius*2+1; k++){
			sum += data[smemPos - kernelRadius + k];
		}
		d_Result[rowStart + writePos] = sum;
#else
		d_Result[rowStart + writePos]  = convolutionRowU<2 * U_KERNEL_RADIUS>(data + smemPos);
#endif
	}
}


__global__ void convolutionColumnGPUFlatU(
	float *d_Result,
	float *d_Data,
	int3 imSize,
	unsigned int kernelRadius,
	int smemStride,
	int gmemStride
){
	extern __shared__ float data[];
	const int         tileStart = IMUL(blockIdx.y, COL_H);
	const int           tileEnd = tileStart + COL_H - 1;
	const int        apronStart = tileStart - kernelRadius;
	const int          apronEnd = tileEnd   + kernelRadius;
	const int    tileEndClamped = min(tileEnd, imSize.y - 1);
	const int apronStartClamped = max(apronStart, 0);
	const int   apronEndClamped = min(apronEnd, imSize.y - 1);
	const int       columnStart = IMUL(blockIdx.x, COL_W) + threadIdx.x;
#ifndef UNROLL_INNER
	float sum ;
#endif
	int smemPos = IMUL(threadIdx.y, COL_W) + threadIdx.x;
	int gmemPos = IMUL(apronStart + threadIdx.y, imSize.z) + columnStart;
	for(int y = apronStart + threadIdx.y; y <= apronEnd; y += blockDim.y){
		data[smemPos] = 
		((y >= apronStartClamped) && (y <= apronEndClamped)) ? 
		d_Data[gmemPos] : 0;
		smemPos += smemStride;
		gmemPos += gmemStride;
	}
	__syncthreads();
	smemPos = IMUL(threadIdx.y + kernelRadius, COL_W) + threadIdx.x;
	gmemPos = IMUL(tileStart + threadIdx.y , imSize.z) + columnStart;
	for(int y = tileStart + threadIdx.y; y <= tileEndClamped; y += blockDim.y){
#ifndef UNROLL_INNER
		sum = 0;
		for(int k = 0; k <2*kernelRadius+1; k++){
			sum += data[smemPos + IMUL(k- kernelRadius, COL_W)];
		}
		d_Result[gmemPos] = sum;
#else
		d_Result[gmemPos] = convolutionColumnU<2 *U_KERNEL_RADIUS>(data + smemPos);
#endif
		smemPos += smemStride;
		gmemPos += gmemStride;
	}

}








/* ======================================== fonction appelantes ======================================== */

void
convolutionSeparableFlatU(float *src , float *dest, float *buff, int3 imSize)
{
	dim3 blockGridRows(iDivUp(imSize.x, ROW_W), imSize.y);
	unsigned int  sizeSharedRow;
	dim3 threadBlockRows(ROW_W +U_KERNEL_RADIUS+iAlignUp(U_KERNEL_RADIUS,ALLIGN_ROW));
	sizeSharedRow =	(U_KERNEL_RADIUS+iAlignUp(U_KERNEL_RADIUS,ALLIGN_ROW) + ROW_W)*sizeof(float);

	convolutionRowGPUFlatU<<<blockGridRows, threadBlockRows,sizeSharedRow>>>(
		buff,
		src,
		U_KERNEL_RADIUS,
		imSize);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );


	dim3 blockGridColumns(iDivUp(imSize.x, COL_W), iDivUp(imSize.y, COL_H));
	size_t sizeSharedCol;
	dim3 threadBlockColumns(COL_W, U_KERNEL_RADIUS);
	sizeSharedCol=COL_W * (U_KERNEL_RADIUS+COL_H+U_KERNEL_RADIUS)*sizeof(float);

	convolutionColumnGPUFlatU<<<blockGridColumns, threadBlockColumns, sizeSharedCol>>>(
		dest,
		buff,
		imSize,
		U_KERNEL_RADIUS,
		COL_W * threadBlockColumns.y,
		imSize.z * threadBlockColumns.y);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

}

