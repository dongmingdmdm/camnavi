#include "hip/hip_runtime.h"
/*
      This file is part of FolkiGpu.

    FolkiGpu is free software: you can redistribute it and/or modify
    it under the terms of the GNU Leeser General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Foobar is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Leeser General Public License
    along with FolkiGpu.  If not, see <http://www.gnu.org/licenses/>.

*/

/*
      FolkiGpu is a demonstration software developed by Aurelien Plyer during
    his phd at Onera (2008-2011). For more information please visit :
      - http://www.onera.fr/dtim-en/gpu-for-image/folkigpu.php
      - http://www.plyer.fr (author homepage)
*/

#include "ConvolutionSeparable.hpp"

__device__ __constant__ float kernel[MAX_KERNEL_RADIUS*2+1];

template<int i> __device__ float convolutionRow(float *data){
    return
        data[MAX_KERNEL_RADIUS - i] * kernel[i]
        + convolutionRow<i - 1>(data);
}

template<> __device__ float convolutionRow<-1>(float *data){
    return 0;
}

template<int i> __device__ float convolutionColumn(float *data){
    return 
        data[(MAX_KERNEL_RADIUS - i) * COL_W] * kernel[i]
        + convolutionColumn<i - 1>(data);
}

template<> __device__ float convolutionColumn<-1>(float *data){
    return 0;
}

__global__ void convolutionRowGPU(
	float *d_Result,
	float *d_Data,
		unsigned int kernelRadius,    
		int3 imSize)
{
	extern __shared__ float data[];
	const int         tileStart = IMUL(blockIdx.x, ROW_W);
	const int           tileEnd = tileStart + ROW_W - 1;
	const int        apronStart = tileStart - kernelRadius;
	const int          apronEnd = tileEnd   + kernelRadius;
	const int    tileEndClamped = min(tileEnd, imSize.x - 1);
	const int apronStartClamped = max(apronStart, 0);
	const int   apronEndClamped = min(apronEnd, imSize.x - 1);
	const int          rowStart = IMUL(blockIdx.y, imSize.z);
	const int apronStartAligned = tileStart - iAlignUp(kernelRadius,ALLIGN_ROW);
	const int           loadPos = apronStartAligned + threadIdx.x;
	const int           smemPos = loadPos - apronStart;
	const int          writePos = tileStart + threadIdx.x;

#ifndef UNROLL_INNER
	float sum ;
#endif

	if(loadPos >= apronStart){
		data[smemPos] = 
			((loadPos >= apronStartClamped) && (loadPos <= apronEndClamped)) ?
			d_Data[rowStart + loadPos] : 0;
	}
	__syncthreads();

	if(writePos <= tileEndClamped){
		const int smemPos = writePos - apronStart;
#ifdef UNROLL_INNER
		d_Result[rowStart + writePos] = convolutionRow<2 * MAX_KERNEL_RADIUS>(data + smemPos);
#else
		sum=0;
		for(int k = 0; k < kernelRadius*2+1; k++){
			sum += data[smemPos - kernelRadius + k] * kernel[k];
				}
		d_Result[rowStart + writePos] = sum;
#endif
	}
}



__global__ void convolutionColumnGPU(
    float *d_Result,
    float *d_Data,
    int3 imSize,
		unsigned int kernelRadius,
    int smemStride,
    int gmemStride
){
    extern __shared__ float data[];
    const int         tileStart = IMUL(blockIdx.y, COL_H);
    const int           tileEnd = tileStart + COL_H - 1;
    const int        apronStart = tileStart - kernelRadius;
    const int          apronEnd = tileEnd   + kernelRadius;
    const int    tileEndClamped = min(tileEnd, imSize.y - 1);
    const int apronStartClamped = max(apronStart, 0);
    const int   apronEndClamped = min(apronEnd, imSize.y - 1);
    const int       columnStart = IMUL(blockIdx.x, COL_W) + threadIdx.x;
    int smemPos = IMUL(threadIdx.y, COL_W) + threadIdx.x;
    int gmemPos = IMUL(apronStart + threadIdx.y, imSize.z) + columnStart;
#ifndef UNROLL_INNER
	float sum ;
#endif

    for(int y = apronStart + threadIdx.y; y <= apronEnd; y += blockDim.y){
        data[smemPos] = 
        ((y >= apronStartClamped) && (y <= apronEndClamped)) ? 
        d_Data[gmemPos] : 0;
        smemPos += smemStride;
        gmemPos += gmemStride;
    }
    __syncthreads();
	smemPos = IMUL(threadIdx.y + kernelRadius, COL_W) + threadIdx.x;
	gmemPos = IMUL(tileStart + threadIdx.y , imSize.z) + columnStart;
	for(int y = tileStart + threadIdx.y; y <= tileEndClamped; y += blockDim.y){
#ifdef UNROLL_INNER
		d_Result[gmemPos] = convolutionColumn<2 * MAX_KERNEL_RADIUS>(data + smemPos);
#else
	sum = 0;
	for(int k = 0; k <2*kernelRadius+1; k++){
		sum += 
			data[smemPos + IMUL(k- kernelRadius, COL_W)] *
			kernel[k];
		}
		d_Result[gmemPos] = sum;
#endif
		smemPos += smemStride;
		gmemPos += gmemStride;
	}

}



void
convolutionSeparable(	float *src , float *dest, float *buff, int3 imSize,
											float * kernelRow, float *kernelCol,unsigned int kernelRowRadius, unsigned int kernelColRadius )
{
	dim3 blockGridRows(iDivUp(imSize.x, ROW_W), imSize.y);
	unsigned int  sizeSharedRow;
	unsigned int  kernelSizeByte;
	dim3 threadBlockRows(ROW_W + kernelRowRadius+iAlignUp(kernelRowRadius,ALLIGN_ROW));
	kernelSizeByte = sizeof(float)*(2*kernelRowRadius+1);

	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(kernel), kernelRow, kernelSizeByte) );
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	sizeSharedRow =	(kernelRowRadius+iAlignUp(kernelRowRadius,ALLIGN_ROW) + ROW_W)*sizeof(float);
	convolutionRowGPU<<<blockGridRows, threadBlockRows,sizeSharedRow>>>(
			buff,
			src,
			kernelRowRadius,
			imSize);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	dim3 blockGridColumns(iDivUp(imSize.x, COL_W), iDivUp(imSize.y, COL_H));
	size_t sizeSharedCol;
	dim3 threadBlockColumns(COL_W, kernelColRadius);
	kernelSizeByte = sizeof(float)*(2*kernelColRadius+1);
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(kernel), kernelCol, kernelSizeByte) );
	sizeSharedCol=COL_W * (kernelColRadius+COL_H+kernelColRadius)*sizeof(float);
	convolutionColumnGPU<<<blockGridColumns, threadBlockColumns, sizeSharedCol>>>(
			dest,
			buff,
			imSize,
			kernelColRadius,
			COL_W * threadBlockColumns.y,
			imSize.z * threadBlockColumns.y);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

}

